#include "hip/hip_runtime.h"
﻿/***
* SLIDING WINDOW SUM
* 
* launch a CUDA kernel that takes a vector of floats with a minimum of 5 elements, 
* applies a sliding window of 2 elements backwards and 2 forwards, calculates the average 
* of those elements and the current element and saves the result in the same index of the 
* current element but in another vector. The maximum size of the vector is the maximum 
* number of threads per block in X.
* 
* @Author: Braulio Solorio
* @Author: Tijash Salamanca
*/
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <ctime>

__global__ void transformToAverageVector(int* vecA, int* vecB, int size);
__host__ void transformToAverageVectorHost(int* vecA, int* vecB, int size);
__host__ void printVector(int* vec, int size, std::string);
__host__ int getMaxThreadNum();
__host__ bool cudaOperationSucced(std::string);

int main() {
	const int MAX_SIZE = getMaxThreadNum();
	if (MAX_SIZE == -1) return 1;

	int vecSize = 1;
	char showVectorsChar = 'y';

	std::cout << "======================== " << std::endl;
	std::cout << "| MIDTERM 1:           | " << std::endl;
	std::cout << "+----------------------| " << std::endl;
	std::cout << "| Duedate: feb 24 2025 | " << std::endl;
	std::cout << "|                      | " << std::endl;
	std::cout << "| Authors: Braulio S.  | " << std::endl;
	std::cout << "|          Tijash S.   | " << std::endl;
	std::cout << "======================== " << std::endl;
	std::cout << "Min vector size: 5" << std::endl;
	std::cout << "Max vector size: " << MAX_SIZE << std::endl;
	std::cout << "\nVector size\n[i]: ";
	std::cin >> vecSize;

	if (vecSize < 5) {
		std::cout << "Size must be in the range [5, " << MAX_SIZE << "]" << std::endl;
		return 1;
	}


	std::cout << "\nShow Final Vectors? (y/n)\n[i]: ";
	std::cin >> showVectorsChar;

	bool showVectors = showVectorsChar != 'n'; 
	

	int* vecA, * vecB;
	int* dev_vecA, * dev_vecB;

	vecA = (int*)malloc(sizeof(int) * vecSize);
	vecB = (int*)malloc(sizeof(int) * vecSize);

	hipMalloc((void**)&dev_vecA, sizeof(int) * vecSize);
	if (!cudaOperationSucced("Cuda Malloc of dev_vecA")) return 1;

	 hipMalloc((void**)&dev_vecB, sizeof(int) * vecSize);
	if (!cudaOperationSucced("Cuda Malloc of dev_vecB")) return 1;

	for (int i = 0; i < vecSize; i++) {
		vecA[i] = i + 1;
	}

	clock_t start = clock();
	transformToAverageVectorHost(vecA, vecB, vecSize);
	clock_t end = clock();
	if (showVectors) {
		std::cout << "CPU" << std::endl;
		printVector(vecA, vecSize, "Vector A");
		printVector(vecB, vecSize, "Vector B");
		std::cout << std::endl;
	}

	double CPU_TIME = ((double)(end - start)) / CLOCKS_PER_SEC;


	dim3 myGrid(1, 1, 1);
	dim3 myBlock(vecSize, 1, 1);


	hipMemcpy(dev_vecA, vecA, sizeof(int) * vecSize, hipMemcpyHostToDevice);
	if (!cudaOperationSucced("Cuda Memcpy of from vecA to dev_vecA")) return 1;

	start = clock();
	transformToAverageVector << <myGrid, myBlock >> > (dev_vecA, dev_vecB, vecSize);
	if (!cudaOperationSucced("transformToAverageVector in device")) return 1;

	hipDeviceSynchronize();
	if (!cudaOperationSucced("Cuda Device Synchronize")) return 1;

	end = clock();
	double GPU_TIME = ((double)(end - start)) / CLOCKS_PER_SEC;

	hipMemcpy(vecA, dev_vecA, sizeof(int) * vecSize, hipMemcpyDeviceToHost);
	if (!cudaOperationSucced("Cuda Memcpy of from dev_vecA to vecA")) return 1;

	hipMemcpy(vecB, dev_vecB, sizeof(int) * vecSize, hipMemcpyDeviceToHost);
	if (!cudaOperationSucced("Cuda Memcpy of from dev_vecb to vecB")) return 1;

	if (showVectors) {
		std::cout << "GPU" << std::endl;
		printVector(vecA, vecSize, "Vector A");
		printVector(vecB, vecSize, "Vector B");
		std::cout << std::endl;
	}

	std::cout << "CPU: " << CPU_TIME << std::endl;
	std::cout << "GPU: " << GPU_TIME << std::endl;

	free(vecA);
	free(vecB);
	hipFree(dev_vecA);
	if (!cudaOperationSucced("Cuda Free of dev_vecA")) return 1;
	hipFree(dev_vecB);
	if (!cudaOperationSucced("Cuda Free of dev_vecB")) return 1;


	return 0;
}

/*
* Transform the values of vecB to the average of a window of 5 numbers from vecA. This function is executed on the device.
* Example:
*	@brief vecA = [1,2,3,4,5].
*	@brief vecB[2] = (1 + 2 + 3 + 4 + 5) / 5 = 3
*	@brief vecB[3] = (2 + 3 + 4 + 5 + 1) / 5 = 3
*   @brief vecB = [3,3,3,3,3,]
* 
* 
* @param vecA - a pointer to the first array of integers
* @param vecB - a pointer to the second array that will be transformed
* @param size - the size of the vectors.
* 
* @returns void
*/
__global__ void transformToAverageVector(int* vecA, int* vecB, int size) {
	int i = (blockDim.x * blockIdx.x + threadIdx.x);
	int idx = size + (i - 2);
	vecB[i] = (vecA[(idx + 0) % size] + vecA[(idx + 1) % size] + vecA[(idx + 2) % size] + vecA[(idx + 3) % size] + vecA[(idx + 4) % size]) / 5;
}

/*
* Transform the values of vecB to the average of a window of 5 numbers from vecA. This function is executed on the host
* Example:
*	@brief vecA = [1,2,3,4,5].
*	@brief vecB[2] = (1 + 2 + 3 + 4 + 5) / 5 = 3
*	@brief vecB[3] = (2 + 3 + 4 + 5 + 1) / 5 = 3
*   @brief vecB = [3,3,3,3,3,]
*
* @param vecA - a pointer to the first array of integers
* @param vecB - a pointer to the second array that will be transformed
* @param size - the size of the vectors.
*
* @returns void
*/
__host__ void transformToAverageVectorHost(int* vecA, int* vecB, int size) {
	for (int i = 0; i < size; i++) {
		int idx = size + (i - 2);
		vecB[i] = (vecA[(idx + 0) % size] + vecA[(idx + 1) % size] + vecA[(idx + 2) % size] + vecA[(idx + 3) % size] + vecA[(idx + 4) % size]) / 5;
	}
}

/*
* Displays a vector in console
* 
* @param vec - the vector to be displayed
* @param size - the size of the vector to be displayed
* @param name - A name tag to the vector that will be displayed in console
*/
__host__ void printVector(int* vec, int size, std::string name = "") {
	std::cout << name << ": ";
	for (int i = 0; i < size; i++) std::cout << vec[i] << " ";
	std::cout << std::endl;
}

/*
* Check the device properties and returns the max number of threads in the X dimension or -1 if an error occurred
* 
* @returns a integer representing the maximun number of threads in the X dimension or -1 if an error occurred
*/
__host__ int getMaxThreadNum() {
	hipDeviceProp_t properties;
	hipDeviceProp_t* properties_ptr = &properties;

	hipGetDeviceProperties(properties_ptr, 0);

	int maxThreads = properties.maxThreadsDim[0];

	if (cudaOperationSucced("Reading device properties")) return maxThreads;

	return -1;
}

/*
* Gets the last cuda Error and if error is not hipSuccess then prints it in console:
* 
* @param action - a description tag to know when the error occurred
* @returns if the cuda error is equal to hipSuccess
*/
__host__ bool cudaOperationSucced(std::string action) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::cout << "[\033[31m" << "ERROR" << "\033[0m]" << " " << hipGetErrorString(error) << ((action != "") ? " @ " : "") << action << std::endl;
		return false;
	}

	return true;
}